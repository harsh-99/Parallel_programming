#include "hip/hip_runtime.h"
#include "headers.h"
/**
 * Host main routine
 */

float check(float *a, float *b){
    float c;
    c = a[0*dimension+1]*b[1*mask_width+0] + a[0*dimension+2]*b[1*mask_width+1] + a[0*dimension+3]*b[1*mask_width+2] + a[1*dimension+1]*b[2*mask_width+0]+ a[1*dimension+2]*b[2*mask_width+1] + a[1*dimension+3]*b[2*mask_width+2]; 
    return c;
}

int main(void)
{
    hipError_t err = hipSuccess;
    // int tile_width = 4;
    // int dimension = 8;
    int numElements = dimension*dimension;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    float *h_A = (float *)malloc(size);

    if (h_A == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    float *h_C = (float *)malloc(size);

    if (h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // int mask_width = 5;
    int num_mask = mask_width*mask_width;
    size_t size1 = num_mask * sizeof(float);

    float *h_B = (float *)malloc(size1);

    if (h_B == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        // h_B[i] = rand()/(float)RAND_MAX;
    }
    for (int i = 0; i < num_mask; ++i)
    {
        h_B[i] = rand()/(float)RAND_MAX;
        // h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size1);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    for (int i=0; i<numElements; i++){
        printf("%f   ", h_A[i]); 
        if ((i+1)%dimension == 0){
            printf("\n");
        }
    }

    for (int i=0; i<num_mask; i++){
        printf("%f   ", h_B[i]); 
        if ((i+1)%mask_width == 0){
            printf("\n");
        }
    }
    
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_B, h_B, size1, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int a = (dimension/tile_width);
    dim3 X1(a,a,1);
    dim3 Y1(tile_width,tile_width,1);
    printf("Cuda kernel launched\n");
    convolution<<<X1,Y1>>>(d_A, d_B, d_C, tile_width, mask_width, dimension);


    // int threadsPerBlock = 1024;
    // int blocksPerGrid =1;
    // printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    // matrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, numElements, dimension);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    for (int i=0; i<numElements; i++){
        printf("%f  ",h_C[i]);
        if ((i+1)%dimension == 0){
            printf("\n");
        }
    }

    printf("Checking one element\n");
    float check_value;
    check_value = check(h_A, h_B); 
    printf("The value checked is %f\n", check_value); 	
    // Verify that the result vector is correct
    // for (int i = 0; i < numElements; ++i)
    // {
    //     if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
    //     {
    //         fprintf(stderr, "Result verification failed at element %d!\n", i);
    //         exit(EXIT_FAILURE);
    //     }
    // }

    // printf("Test PASSED\n");

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}

