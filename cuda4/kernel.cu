
#include <hip/hip_runtime.h>
__global__ void 
convolution(float *N , float *M, float *P , int Tile_Size, int Mask_Width , int Width)
{

int k = blockIdx.x*blockDim.x + threadIdx.x;
int l = blockIdx.x*blockDim.x + threadIdx.x;

__shared__ float N_ds[4][4];

int n = Mask_Width/2;
int halo_index_left = (blockIdx.x - 1) * blockDim.x + threadIdx.x;
int halo_index_top = (blockIdx.y - 1) * blockDim.y + threadIdx.y;
int halo_index_right = (blockIdx.x +1) * blockDim.x + threadIdx.x;
int halo_index_bottom = (blockIdx.y + 1) * blockDim.y + threadIdx.y;

if(threadIdx.x >= blockDim.x - n && threadIdx.y >= blockDim.y - n) N_ds[threadIdx.y -(blockDim.y - n)][threadIdx.x -(blockDim.x - n)] = (halo_index_left < 0 || halo_index_top < 0)?0:N[halo_index_top* Width + halo_index_left];
else if(threadIdx.x < n && threadIdx.y >= blockDim.y - n) N_ds[threadIdx.y -(blockDim.y - n)][n + blockDim.x + threadIdx.x] = (halo_index_right >= Width || halo_index_top < 0)?0:N[halo_index_top*Width + halo_index_right];
else if(threadIdx.y < n && threadIdx.x >= blockDim.x - n) N_ds[n + blockDim.y + threadIdx.y][threadIdx.x -(blockDim.x - n)] = (halo_index_bottom >= Width || halo_index_left < 0)?0:N[halo_index_bottom*Width + halo_index_left];
else if(threadIdx.x < n && threadIdx.y < n) N_ds[n + blockDim.y + threadIdx.y][n + blockDim.x + threadIdx.x] = (halo_index_right >= Width || halo_index_bottom >= Width)?0:N[halo_index_bottom*Width + halo_index_right];
else if(threadIdx.y < n) N_ds[n + blockDim.y + threadIdx.y][n + threadIdx.x] = (halo_index_bottom >=Width)?0:N[(halo_index_bottom*Width) + (blockIdx.x * blockDim.x + threadIdx.x)];
else if(threadIdx.x < n) N_ds[n + threadIdx.y][n + blockDim.x + threadIdx.x] = (halo_index_right >=Width)?0:N[(blockDim.y * blockIdx.y + threadIdx.y)*Width + (halo_index_right)];
else if(threadIdx.y >= blockDim.y - n) N_ds[threadIdx.y -(blockDim.y - n)][n+threadIdx.x] = (halo_index_top < 0)?0:N[(halo_index_top*Width) + (blockDim.x*blockIdx.x + threadIdx.x)];
else if(threadIdx.x >= blockDim.x -n) N_ds[n+threadIdx.y][threadIdx.x -(blockDim.x - n)] = (halo_index_left<0)?0:N[(blockDim.y * blockIdx.y + threadIdx.y)*Width + halo_index_left];
else N_ds[n + threadIdx.y][n + threadIdx.x]= N[(blockIdx.y * blockDim.y + threadIdx.y)*Width + (blockIdx.x * blockDim.x + threadIdx.x)];

__syncthreads();

float Pvalue = 0;
for(int i =0; i < Mask_Width; i++){
    for(int j =0; j < Mask_Width ; j++) {
        Pvalue += N_ds[threadIdx.y + i][threadIdx.x + j] * M[(i*Mask_Width) + j];
    }
}

P[(l*Width) + k] = Pvalue;
}

