
#include <hip/hip_runtime.h>
__global__ void 
convolution(float *N , float *M, float *P , int Tile_Size, int Mask_Width , int Width)
{

int k = blockIdx.x * blockDim.x + threadIdx.x;
int l = blockIdx.y * blockDim.y + threadIdx.y;

__shared__ float N_ds[8][8];

int n = Mask_Width/2;
int halo_index_left = (blockIdx.x - 1) * blockDim.x + threadIdx.x;
int halo_index_top = (blockIdx.y - 1) * blockDim.y + threadIdx.y;
int halo_index_right = (blockIdx.x +1) * blockDim.x + threadIdx.x;
int halo_index_bottom = (blockIdx.y + 1) * blockDim.y + threadIdx.y;

N_ds[n + threadIdx.y][n + threadIdx.x]= N[(blockIdx.y * blockDim.y + threadIdx.y)*Width + (blockIdx.x * blockDim.x + threadIdx.x)];
if(threadIdx.x >= blockDim.x-n && threadIdx.y >= blockDim.y - n) {
    N_ds[threadIdx.y -(blockDim.y - n)][threadIdx.x -(blockDim.x - n)] = (halo_index_left < 0 || halo_index_top < 0)?0:N[halo_index_top* Width + halo_index_left];
    N_ds[threadIdx.y -(blockDim.y - n)][n+threadIdx.x] = (halo_index_top<0)?0:N[halo_index_top*Width + (blockDim.x*blockIdx.x + threadIdx.x)];
    N_ds[n+threadIdx.y][threadIdx.x -(blockDim.x - n)] = (halo_index_left<0)?0:N[(blockDim.y*blockIdx.y + threadIdx.y)*Width + halo_index_left];
}
if(threadIdx.x < n && threadIdx.y >= blockDim.y - n){ 
    N_ds[threadIdx.y -(blockDim.y - n)][n + blockDim.x + threadIdx.x] = (halo_index_right >= Width || halo_index_top < 0)?0:N[halo_index_top*Width + halo_index_right];
    N_ds[threadIdx.y -(blockDim.y - n)][n+threadIdx.x] = (halo_index_top<0)?0:N[halo_index_top*Width + (blockDim.x*blockIdx.x + threadIdx.x)];
    N_ds[n+threadIdx.y][n + blockDim.x + threadIdx.x] = (halo_index_right >= Width)?0:N[(blockDim.y*blockIdx.y + threadIdx.y)*Width + halo_index_right];
}
if(threadIdx.y < n && threadIdx.x >= blockDim.x - n){ 
    N_ds[n + blockDim.y + threadIdx.y][threadIdx.x -(blockDim.x - n)] = (halo_index_bottom >= Width || halo_index_left < 0)?0:N[halo_index_bottom*Width + halo_index_left];
    N_ds[n + blockDim.y + threadIdx.y][n+threadIdx.x] = (halo_index_bottom >= Width)?0:N[halo_index_bottom*Width + (blockDim.x*blockIdx.x + threadIdx.x)];
    N_ds[n+threadIdx.y][threadIdx.x -(blockDim.x - n)] = (halo_index_left < 0)?0:N[(blockDim.y*blockIdx.y + threadIdx.y)*Width + halo_index_left];
}
if(threadIdx.x < n && threadIdx.y < n){ 
    N_ds[n + blockDim.y + threadIdx.y][n + blockDim.x + threadIdx.x] = (halo_index_right >= Width || halo_index_bottom >= Width)?0:N[halo_index_bottom*Width + halo_index_right];
    N_ds[n + blockDim.y + threadIdx.y][n+threadIdx.x] = (halo_index_bottom >= Width)?0:N[halo_index_bottom*Width + (blockDim.x*blockIdx.x + threadIdx.x)];
    N_ds[n+threadIdx.y][n + blockDim.x + threadIdx.x] = (halo_index_right >=Width)?0:N[(blockDim.y*blockIdx.y + threadIdx.y)*Width + halo_index_right];
}
if(threadIdx.y < n && threadIdx.x > n && threadIdx.x <= blockDim.x-n) N_ds[n + blockDim.y + threadIdx.y][n + threadIdx.x] = (halo_index_bottom >=Width)?0:N[(halo_index_bottom*Width) + (blockIdx.x * blockDim.x + threadIdx.x)];
if(threadIdx.x < n && threadIdx.y > n && threadIdx.y <= blockDim.y-n) N_ds[n + threadIdx.y][n + blockDim.x + threadIdx.x] = (halo_index_right >=Width)?0:N[(blockDim.y * blockIdx.y + threadIdx.y)*Width + (halo_index_right)];
if(threadIdx.y >= blockDim.y - n && threadIdx.x > n && threadIdx.x <= blockDim.x-n) N_ds[threadIdx.y -(blockDim.y - n)][n+threadIdx.x] = (halo_index_top < 0)?0:N[(halo_index_top*Width) + (blockDim.x*blockIdx.x + threadIdx.x)];
if(threadIdx.x >= blockDim.x - n && threadIdx.y > n && threadIdx.y <= blockDim.y-n) N_ds[n+threadIdx.y][threadIdx.x -(blockDim.x - n)] = (halo_index_left < 0)?0:N[(blockDim.y * blockIdx.y + threadIdx.y)*Width + halo_index_left];


__syncthreads();

float Pvalue = 0;
for(int i =0; i < Mask_Width; i++){
    for(int j =0; j < Mask_Width ; j++) {
        Pvalue += N_ds[threadIdx.y + i][threadIdx.x + j] * M[(i*Mask_Width) + j];
    }
}
P[(l*Width) + k] = Pvalue;
}
