
#include <hip/hip_runtime.h>
__global__ void
matrix(float *A, int numElements, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    float a; 
    if (i < n && j< n && i!= (n-1) && i%2==0)
    {
        	a = A[j*n + i];
        	A[j*n + i] = A[j*n + i + 1];
        	A[j*n + i +1] = a;	 
    }
    if (i<n&& j<n && i<j){
        A[i*n+j] = A[j*n + i];
    }	
    
}

