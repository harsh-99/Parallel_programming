
#include <hip/hip_runtime.h>
__global__ void
process_kernel1(const float *input1,const float *input2, float *output, int datasize){
	int blockNum = blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*(gridDim.x)+blockIdx.x;
	int threadNum = threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*(blockDim.x)+threadIdx.x;
	int i = blockNum*(blockDim.x* blockDim.y * blockDim.z)+ threadNum;

	output[i] = sin(input1[i]) + cos(input2[i]);


}

__global__ void
process_kernel2(float *input1, float *output, int datasize){
	int blockNum = blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*(gridDim.x)+blockIdx.x;
	int threadNum = threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*(blockDim.x)+threadIdx.x;
	int i = blockNum*(blockDim.x* blockDim.y * blockDim.z)+ threadNum;

	output[i] = log(input1[i]);
	
}

__global__ void
process_kernel3(float *input1, float *output, int datasize){
	int blockNum = blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*(gridDim.x)+blockIdx.x;
	int threadNum = threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*(blockDim.x)+threadIdx.x;
	int i = blockNum*(blockDim.x* blockDim.y * blockDim.z)+ threadNum;

	output[i] = sqrt(input1[i]);
	
}
